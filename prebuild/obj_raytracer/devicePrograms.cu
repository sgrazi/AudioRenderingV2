#include "hip/hip_runtime.h"
#include <stdio.h>
#include <optix_device.h>
#include <optix.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <gdt/math/vec.h>
#include <glm/glm.hpp>
#include "LaunchParams.h"
#include "PRD.h"

#define SPEED_OF_SOUND 343 // grabbed from Cameelo/AudioRendering

/*! launch parameters in constant memory, filled in by optix upon
      optixLaunch (this gets filled in from the buffer we pass to
      optixLaunch) */
extern "C" __constant__ LaunchParams optixLaunchParams;

// for this simple example, we have a single ray type
// RAY_TYPE_COUNT does not have an explicit value assigned, the compiler automatically assigns it a value one greater than the previous enumerator
enum
{
    SURFACE_RAY_TYPE = 0,
    RAY_TYPE_COUNT
};

static __forceinline__ __device__ void *unpackPointer(uint32_t i0, uint32_t i1)
{
    const uint64_t uptr = static_cast<uint64_t>(i0) << 32 | i1;
    void *ptr = reinterpret_cast<void *>(uptr);
    return ptr;
}

static __forceinline__ __device__ void packPointer(void *ptr, uint32_t &i0, uint32_t &i1)
{
    const uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}

template <typename T>
static __forceinline__ __device__ T *getPRD()
{
    const uint32_t u0 = optixGetPayload_0();
    const uint32_t u1 = optixGetPayload_1();
    return reinterpret_cast<T *>(unpackPointer(u0, u1));
}

//------------------------------------------------------------------------------
// closest hit and anyhit programs for radiance-type rays.
//
// Note eventually we will have to create one pair of those for each
// ray type and each geometry type we want to render; but this
// simple example doesn't use any actual geometries yet, so we only
// create a single, dummy, set of them (we do have to have at least
// one group of them to set up the SBT)
//------------------------------------------------------------------------------

extern "C" __global__ void __closesthit__radiance()
{

    const TriangleMeshSBTData &sbtData = *(const TriangleMeshSBTData *)optixGetSbtDataPointer();
    const float3 wrd = optixGetWorldRayDirection();
    const glm::vec3 rayDir = glm::vec3(wrd.x, wrd.y, wrd.z);
    PRD &prd = *(PRD *)getPRD<PRD>();

    switch (sbtData.mat)
    {
    case 0:
        // receptor
        const glm::vec3 dist_vec = sbtData.pos - prd.curr_position;
        const float distance = fabs(dot(dist_vec, prd.direction));
        prd.distance += distance;

        float *histogram = optixLaunchParams.histogram;
        float elapsed_time = prd.distance / SPEED_OF_SOUND;
        int array_pos = round(elapsed_time * optixLaunchParams.sample_rate);
        if (array_pos < optixLaunchParams.histogram_length)
            histogram[array_pos] += prd.remaining_factor;
        break;
    default:
        // material
        const int primID = optixGetPrimitiveIndex();
        const glm::ivec3 index = sbtData.index[primID];
        const glm::vec3 &A = sbtData.vertex[index.x];
        const glm::vec3 &B = sbtData.vertex[index.y];
        const glm::vec3 &C = sbtData.vertex[index.z];
        const glm::vec3 Ng = normalize(cross(B - A, C - A));
        prd.direction = prd.direction - 2.0f * (prd.direction * Ng) * Ng;

        prd.curr_position = sbtData.pos;

		float dist_traveled = optixGetRayTmax(); // returns the current path segment distance
        prd.distance += dist_traveled;

        // TO DO, is sbtData.mat the id of the material or the name?
        //uint32_t mat = sbtData.mat;
        //auto result = thrust::find_if(optixLaunchParams.absorption.begin(), optixLaunchParams.absorption.end(),
        //                          [mat] (const Material& m) {
        //                              return m.id == mat;
        //                          });
        //if (result != optixLaunchParams.absorption.end()) {
        //    prd.remaining_factor *= result.base()->ac_absorption;
        //}
        //else {
        //    // material not found
        //    prd.remaining_factor *= 0;
        //}

        prd.remaining_factor *= 0.25;
        
        prd.recursion_depth++;
    }
}

extern "C" __global__ void __anyhit__radiance()
{ /*! TO DO probably */
}

extern "C" __global__ void __miss__radiance()
{
    PRD &prd = *(PRD *)getPRD<PRD>();
    prd.recursion_depth = -1;
}

extern "C" __global__ void __raygen__renderFrame()
{
    
    // TODO, check if dimensions are three dimensional
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;
    const int iz = optixGetLaunchIndex().z;
    const int x_rays = optixGetLaunchDimensions().x; 
    const int y_rays = optixGetLaunchDimensions().y;
    const int z_rays = optixGetLaunchDimensions().z;

    const auto &camera = optixLaunchParams.camera;

    // the values we store the PRD pointer in:
    // Note: Payload Reference Data and represents the data structure used to pass information between shaders during the ray tracing process
    PRD prd;
    uint32_t u0, u1;
    packPointer(&prd, u0, u1);
    prd.remaining_factor = 1.0f;
    prd.distance = 0;
    prd.curr_position = optixLaunchParams.origin_pos;
    prd.recursion_depth = 0;
    
    // TODO distribution of rays should be uniform, to be tested
    float offset = static_cast<float>(ix + iy * x_rays + iz * y_rays * x_rays) / static_cast<float>(x_rays * y_rays * z_rays);
    double theta = 2 * M_PI * offset;
    double phi = acos(1 - 2 * offset);
    double dx = sin(phi) * cos(theta);
    double dy = sin(phi) * sin(theta);
    double dz = cos(phi);
    prd.direction = {dx, dy, dz};

    int i = 0;
    // pack data into payload
    while (prd.distance < optixLaunchParams.dist_thres &&
           prd.remaining_factor > optixLaunchParams.energy_thres &&
           prd.recursion_depth >= 0 &&
           i < 10000) // por las dudas le pongo un tope
    {
        i++;
        gdt::vec3f rayOrigin(camera.position.x, camera.position.y, camera.position.z);
        gdt::vec3f rayDir(prd.direction.x, prd.direction.y, prd.direction.z);
        optixTrace(optixLaunchParams.traversable,
                   rayOrigin,
                   rayDir,
                   0.f,   // tmin
                   1e20f, // tmax
                   0.0f,  // rayTime
                   OptixVisibilityMask(255),
                   OPTIX_RAY_FLAG_DISABLE_ANYHIT, // OPTIX_RAY_FLAG_NONE,
                   SURFACE_RAY_TYPE,              // SBT offset
                   RAY_TYPE_COUNT,                // SBT stride
                   SURFACE_RAY_TYPE,              // missSBTIndex
                   u0, u1);
    }
}