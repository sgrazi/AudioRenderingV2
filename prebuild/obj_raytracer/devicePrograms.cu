#include "hip/hip_runtime.h"
#include <stdio.h>
#include <optix_device.h>
#include <optix.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <gdt/math/vec.h>
#include <glm/glm.hpp>
#include "LaunchParams.h"
#include "PRD.h"

#define SPEED_OF_SOUND 343 // grabbed from Cameelo/AudioRendering

/*! launch parameters in constant memory, filled in by optix upon
      optixLaunch (this gets filled in from the buffer we pass to
      optixLaunch) */
extern "C" __constant__ LaunchParams optixLaunchParams;

// for this simple example, we have a single ray type
// RAY_TYPE_COUNT does not have an explicit value assigned, the compiler automatically assigns it a value one greater than the previous enumerator
enum
{
    SURFACE_RAY_TYPE = 0,
    RAY_TYPE_COUNT
};

static __forceinline__ __device__ void *unpackPointer(uint32_t i0, uint32_t i1)
{
    const uint64_t uptr = static_cast<uint64_t>(i0) << 32 | i1;
    void *ptr = reinterpret_cast<void *>(uptr);
    return ptr;
}

static __forceinline__ __device__ void packPointer(void *ptr, uint32_t &i0, uint32_t &i1)
{
    const uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}

template <typename T>
static __forceinline__ __device__ T *getPRD()
{
    const uint32_t u0 = optixGetPayload_0();
    const uint32_t u1 = optixGetPayload_1();
    return reinterpret_cast<T *>(unpackPointer(u0, u1));
}

//------------------------------------------------------------------------------
// closest hit and anyhit programs for radiance-type rays.
//
// Note eventually we will have to create one pair of those for each
// ray type and each geometry type we want to render; but this
// simple example doesn't use any actual geometries yet, so we only
// create a single, dummy, set of them (we do have to have at least
// one group of them to set up the SBT)
//------------------------------------------------------------------------------

extern "C" __global__ void __closesthit__radiance()
{
    const TriangleMeshSBTData &sbtData = *(const TriangleMeshSBTData *)optixGetSbtDataPointer();
    const float3 wrd = optixGetWorldRayDirection();
    const glm::vec3 rayDir = glm::vec3(wrd.x, wrd.y, wrd.z);
    PRD &prd = *(PRD *)getPRD<PRD>();

    const int primID = optixGetPrimitiveIndex();
    const glm::ivec3 index = sbtData.index[primID];
    const glm::vec3& A = sbtData.vertex[index.x];
    const glm::vec3& B = sbtData.vertex[index.y];
    const glm::vec3& C = sbtData.vertex[index.z];
    const glm::vec3 Ng = normalize(cross(B - A, C - A));
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;
    glm::vec3 P = (1.f - u - v) * A + u * B + v * C;

    switch (sbtData.mat_absorption < 0) // we identify the receiver with a negative absorption
    {
    case true:
        // printf("HIT RECEIVER at: %f,%f,%f...\n", P.x, P.y, P.z);
        prd.distance += distance(P,prd.prev_position);
        float elapsed_time = prd.distance / SPEED_OF_SOUND;
        int array_pos = round(elapsed_time * optixLaunchParams.sample_rate);
        float *ir = optixLaunchParams.ir;
        if (array_pos < optixLaunchParams.ir_length) {
            ir[array_pos] += prd.remaining_factor;
        }
        break;
    case false:
        //printf("HIT MATERIAL at: %f,%f,%f...\n", P.x, P.y, P.z);
        // material
        prd.direction = prd.direction - 2.0f * (prd.direction * Ng) * Ng;
		float dist_traveled = optixGetRayTmax(); // returns the current path segment distance
        prd.distance += dist_traveled;
        prd.remaining_factor *= (1 - sbtData.mat_absorption);
        prd.recursion_depth++;
        break;
    default:
        // ERROR
    }
    prd.prev_position = P;
}

extern "C" __global__ void __anyhit__radiance()
{ 
}

extern "C" __global__ void __miss__radiance()
{
    PRD &prd = *(PRD *)getPRD<PRD>();
    prd.recursion_depth = -1;
}

extern "C" __global__ void __raygen__renderFrame()
{
    
    // TODO, check if dimensions are three dimensional
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;
    const int iz = optixGetLaunchIndex().z;
    const int x_rays = optixGetLaunchDimensions().x; 
    const int y_rays = optixGetLaunchDimensions().y;
    const int z_rays = optixGetLaunchDimensions().z;

    // the values we store the PRD pointer in:
    // Note: Payload Reference Data and represents the data structure used to pass information between shaders during the ray tracing process
    PRD prd;
    uint32_t u0, u1;
    packPointer(&prd, u0, u1);
    prd.remaining_factor = (optixLaunchParams.BASE_POWER)/(x_rays*y_rays*z_rays);
    prd.distance = 0;
    prd.prev_position = optixLaunchParams.emitter_position;
    prd.recursion_depth = 0;

    double dx = (ix * (2.0 / (x_rays - 1)) - 1.0);
    double dy = (iy * (2.0 / (y_rays - 1)) - 1.0);
    double dz = (iz * (2.0 / (z_rays - 1)) - 1.0);
    // it is bound to happen that some threads have (0,0,0) as their vector
    if (dx != 0.0 || dy != 0.0 || dz != 0.0) {
        double length = std::sqrt(dx * dx + dy * dy + dz * dz);
        dx /= length;
        dy /= length;
        dz /= length;

        // printf("sending to %f,%f,%f...\n", dx, dy, dz);
        prd.direction = {dx, dy, dz};
        int i = 0;
        while (prd.distance < optixLaunchParams.dist_thres &&
               prd.remaining_factor > optixLaunchParams.energy_thres &&
               prd.recursion_depth >= 0 &&
               i < 60) // por las dudas le pongo un tope
        {
            i++;
            gdt::vec3f rayOrigin(prd.prev_position.x, prd.prev_position.y, prd.prev_position.z);
            gdt::vec3f rayDir(prd.direction.x, prd.direction.y, prd.direction.z);
            optixTrace(optixLaunchParams.traversable,
                       rayOrigin,
                       rayDir,
                       0.f,   // tmin
                       1e20f, // tmax
                       0.0f,  // rayTime
                       OptixVisibilityMask(255),
                       OPTIX_RAY_FLAG_DISABLE_ANYHIT, // OPTIX_RAY_FLAG_NONE,
                       SURFACE_RAY_TYPE,              // SBT offset
                       RAY_TYPE_COUNT,                // SBT stride
                       SURFACE_RAY_TYPE,              // missSBTIndex
                       u0, u1);
        }
    }
}