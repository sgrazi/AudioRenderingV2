#include "hip/hip_runtime.h"
#include "./kernels.cuh"
#define CUDA_CHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void CHCK_CUFFT_RES(hipfftResult_t res){
    if (res != 0)
    {
        fprintf(stderr, "CHCK_CUFFT_RES: %d\n", res);
        if (abort) exit(res);
    }
}

inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void fillZeros(float *buf, int size)
{
    int thread = blockDim.x * blockIdx.x + threadIdx.x;
    if (thread < size)
        buf[thread] = 0.f;
}

void fillWithZeroesKernel(float *buf, int size)
{
    int numThreads = 256;
    int numBlocks;
    if (size % numThreads != 0) {
        numBlocks = (size / numThreads) + 1;
    } else {
        numBlocks = size / numThreads;
    }
    fillZeros<<<numBlocks, numThreads>>>(buf, size);
}

__global__ void vectorMultiply(float* a, float* b, float* c, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        c[index] = a[index] * b[index];
    }
}

__global__ void convolute_toeplitz_lower_matrix(float* samples, float* IR, size_t ir_size, float* outputBuffer){
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // where in the "matrix" are we
    int row, col;
    for (int i = 1; i <= ir_size; i++) {
        if (index < i) {
            row = i - 1;
            col = index;
            break;
        }
        index -= i;
    }
    
    /*
    because cell with col = row has I0
    then cell with col = row - 1 has I1 (because we moved column one to the left)
    then cell with col = row - x has Ix
    then row - col = x
    */
    int ir_index = row - col;
    atomicAdd(&outputBuffer[col], samples[row] * IR[ir_index]);
}

__global__ void convolute_toeplitz_lower_matrix_2d(float* samples, float* IR, size_t ir_size, float* outputBuffer) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x <= y && (x < ir_size && y < ir_size)) {
        atomicAdd(&outputBuffer[y*2], samples[x] * IR[y - x]);
        atomicAdd(&outputBuffer[(y*2) - 1], samples[x] * IR[y - x]);
    }
}

__global__ void convolute_toeplitz_vectors(float* samples, float* IR, size_t ir_size, float* outputBuffer){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int samples_offset = index / ir_size;
    int ir_index = index % ir_size;
    /*
    for this part of the convolution samples start at ir_len, thats why we sum ir_len to the samples index
    for each complete multiplication we (the whole IR vector * a subvector of samples) we need to continue with a new subvector
    this new subvector is the samples vector moved one unit forward, thats why we add a samples_offset to the samples index
    */
    atomicAdd(&outputBuffer[ir_size + samples_offset], samples[ir_size + samples_offset] * IR[ir_size - 1 - ir_index]);
}

void convolute_toeplitz_in_gpu(float* samples, float* IR, int ir_len, float* outputBuffer){
    //printf("ir_len: %d", ir_len);
    const int threadsPerBlock = 256;

    // first part, lower matrix multiplication
    /*
    cantidad operaciones = (ir_len * (ir_len + 1)) / 2
    esto es porque en la multiplicacion de la matriz triangular inferior
    la primera fila tiene una operacion, la segunda dos, y la n-esima (hasta el tope que es ir_len)
    equivale a Σ(i) de i=0 a i=n la cual se resuelve con ((n * (n + 1)) / 2)
    */
    int blocksPerGrid;
    if (((ir_len * (ir_len + 1)) / 2) < threadsPerBlock)
        blocksPerGrid = 1;
    else
        blocksPerGrid = (((ir_len * (ir_len + 1)) / 2) / threadsPerBlock) + 1;
    //convolute_toeplitz_lower_matrix<<<blocksPerGrid, threadsPerBlock>>>(samples, IR, ir_len, outputBuffer);
    dim3 threadsPorBlocks(32,32);
    int aaa = (ir_len / 32) + 1;
    dim3 numBlocks(aaa, aaa);
    convolute_toeplitz_lower_matrix_2d << <numBlocks, threadsPorBlocks >> > (samples, IR, ir_len, outputBuffer);

    // second part, vector multiplication
    size_t samples_size = sizeof(samples) / sizeof(float);
    /*
    ya procesamos ir_len celdas del output
    quedan (samples_size - ir_len) celdas restantes
    cada celda tiene ir_len multiplicaciones
    */
    //blocksPerGrid = ((samples_size - ir_len) * ir_len) / threadsPerBlock;
    //convolute_toeplitz_vectors<<<blocksPerGrid, threadsPerBlock>>>(samples, IR, ir_len, outputBuffer); // todo se precisa un offset
}

__global__ void load_complex_vector(hipfftComplex* complex_data, float* real_vector, unsigned int vector_len) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < vector_len){
        complex_data[idx].x = real_vector[idx];
        complex_data[idx].y = 0.0f;
    }
}

__global__ void load_sample_segment(int second, unsigned int sampleRate, unsigned int segment_size_in_seconds, float* segment, float* samples) {
    // loads one second of samples and (segment_size_in_seconds - 1) seconds of zeros
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int sampleIndex = idx + second * sampleRate;

    if (idx < sampleRate * segment_size_in_seconds){
        if (idx < sampleRate) {
            segment[idx] = samples[sampleIndex];
        }
        else {
            segment[idx] = 0.0f;
        }
    }
}

__global__ void multiply_samples_segment_and_ir(int second, unsigned int sampleRate, unsigned int segment_size_in_seconds, hipfftComplex* sampleData, hipfftComplex* IRData) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < sampleRate * segment_size_in_seconds){
        // (a + ib) (c + id) = (ac – bd) + i(ad + bc)
        hipfftComplex temp = sampleData[idx];
        sampleData[idx].x = temp.x * IRData[idx].x - temp.y * IRData[idx].y;
        sampleData[idx].y = temp.x * IRData[idx].y + temp.y * IRData[idx].x;
    }
}

__global__ void add_segment_to_result_buffer(int second, int sampleRate, int segmentLen, float* segment, float* output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // printf("SAMPLE SEGMENT: %d\n", segment);
    if (index < segmentLen) {
        int outputIndex = second * sampleRate + index;
        atomicAdd(&output[outputIndex], segment[index]);
    }
}

__global__ void isAllZeros(float* arr, int size, int* result) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        if (arr[index] != 0.0f) {
            atomicExch(result, 1); // Set result to 1 if any element is not zero
        }
    }
}

// CUDA kernel to check each element if it's zero (complex numbers)
__global__ void isAllZerosComplex(hipfftComplex* arr, int size, int* result) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        if (arr[index].x != 0.0f || arr[index].y != 0.0f) {
            atomicExch(result, 1); // Set result to 1 if any element is not zero
        }
    }
}

bool checkArrayZero(float* IR, unsigned int ir_len) {
    int* d_result; // Device result
    int h_result = 0; // Host result

    // Calculate grid and block sizes
    int blockSize = 256; // Number of threads per block
    int numBlocks = (ir_len + blockSize - 1) / blockSize;
    hipMalloc(&d_result, sizeof(int));
    hipMemcpy(d_result, &h_result, sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel
    isAllZeros << <numBlocks, blockSize >> > (IR, ir_len, d_result);

    // Copy the result back to host
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_result);

    // Check the result
    if (h_result == 0) {
        printf("All elements are zero.\n");
        return true;
    }
    else {
        printf("There is at least one element that is not zero.\n");
        return false;
    }

    
}

void convolute_fourier_in_gpu(float* samples, float* IR, unsigned int samples_len, unsigned int sample_rate, unsigned int ir_len, float* outputBuffer) {
    const int threadsPerBlock = 256;
    int blocks;
    const int batchSize = 1; // Number of batches 
    const int secondsToProcess = samples_len / sample_rate;
    const int segment_size_in_seconds = 2;
    
    //int* d_result; // Device result
    //int h_result = 0; // Host result

    //// Calculate grid and block sizes
    //int blockSize = 256; // Number of threads per block
    //int numBlocks = (ir_len + blockSize - 1) / blockSize;
    //hipMalloc(&d_result, sizeof(int));
    //hipMemcpy(d_result, &h_result, sizeof(int), hipMemcpyHostToDevice);

    //// Launch the kernel
    //isAllZeros << <numBlocks, blockSize >> > (IR, ir_len, d_result);

    //// Copy the result back to host
    //hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    //// Check the result
    //if (h_result == 0) {
    //    printf("All elements are zero.\n");
    //}
    //else {
    //    printf("There is at least one element that is not zero.\n");
    //}

    //// Free GPU memory
    //hipFree(d_result);

    // Allocate device memory for samples
    float* sampleSegment;
    int segmentSize = sample_rate * segment_size_in_seconds * sizeof(float);
    hipMalloc((void**)&sampleSegment, segmentSize);
    hipfftComplex* segmentData;
    hipMalloc((void**)&segmentData, sample_rate * segment_size_in_seconds * sizeof(hipfftComplex));
    // Allocate device memory for IR
    hipfftComplex* IRData;
    hipMalloc((void**)&IRData, ir_len * sizeof(hipfftComplex));
    // Allocate device memory for output
    hipfftComplex* resultData;
    hipMalloc((void**)&resultData, samples_len * sizeof(hipfftComplex));
    // Set up FFT plans
    hipfftHandle segmentPlan;
    hipfftPlan1d(&segmentPlan, sample_rate * segment_size_in_seconds, HIPFFT_R2C, batchSize);
    hipfftHandle IRPlan;
    hipfftPlan1d(&IRPlan, ir_len, HIPFFT_R2C, batchSize);
    // Invert result
    hipfftHandle inversePlan;
    hipfftPlan1d(&inversePlan, sample_rate * segment_size_in_seconds, HIPFFT_C2R, batchSize);
    // Do FFT on IR, which will be reused a lot
    blocks = (ir_len / threadsPerBlock) + 1;
    hipDeviceSynchronize();
    CHCK_CUFFT_RES(hipfftExecR2C(IRPlan, IR, IRData));

    //int* d_result; // Device result
    //int h_result = 0; // Host result

    //// Allocate memory on the GPU
    //hipMalloc(&d_result, sizeof(int));

    //// Copy the host array to the device (GPU)
    //hipMemcpy(d_result, &h_result, sizeof(int), hipMemcpyHostToDevice);

    //// Calculate grid and block sizes
    //int blockSize = 256; // Number of threads per block
    //int numBlocks = (ir_len + blockSize - 1) / blockSize;

    //// Launch the kernel
    //isAllZerosComplex << <numBlocks, blockSize >> > (IRData, ir_len, d_result);

    //// Copy the result back to host
    //hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    //// Check the result
    //if (h_result == 0) {
    //    printf("COMPLEX = All elements are zero.\n");
    //}
    //else {
    //    printf("COMPLEX = There is at least one element that is not zero.\n");
    //}

    //hipFree(d_result);


    // Finally convolute, second by second
    /*
    Basically we take each segment and we prolong it with 0's
    Then we do FFT and sum each segment into the total
    Then we invert the total
    Full algorithm can be found on https://www.dspguide.com/ch18/2.htm
    */
    blocks = ((sample_rate * segment_size_in_seconds) / threadsPerBlock) + 1;
    for (int second = 0; second < secondsToProcess; second++) {
        // First second is samples, rest is 0's (this is why we do seconds + 1 as the upper limit)
        load_sample_segment << <blocks, threadsPerBlock >> > (second, sample_rate, segment_size_in_seconds, sampleSegment, samples);
        hipDeviceSynchronize();
        // FFT on the loaded segment, and convolute it with IR
        CHCK_CUFFT_RES(hipfftExecR2C(segmentPlan, sampleSegment, segmentData));
        multiply_samples_segment_and_ir << <blocks, threadsPerBlock >> > (second, sample_rate, segment_size_in_seconds, segmentData, IRData);
        hipDeviceSynchronize();
        // Inverse on the result and save it to buffer
        CHCK_CUFFT_RES(hipfftExecC2R(inversePlan, segmentData, sampleSegment));
        blocks = (segmentSize + threadsPerBlock - 1) / threadsPerBlock;
        add_segment_to_result_buffer<<<blocks, threadsPerBlock>>>(second, sample_rate, sample_rate * segment_size_in_seconds, sampleSegment, outputBuffer);
        hipDeviceSynchronize();
    }
    int* d_result; // Device result
    int h_result = 0; // Host result

    // Allocate memory on the GPU
    hipMalloc(&d_result, sizeof(int));

    // Copy the host array to the device (GPU)
    hipMemcpy(d_result, &h_result, sizeof(int), hipMemcpyHostToDevice);

    // Calculate grid and block sizes
    int blockSize = 256; // Number of threads per block
    int numBlocks = (ir_len + blockSize - 1) / blockSize;

    // Launch the kernel
    isAllZeros << <numBlocks, blockSize >> > (outputBuffer, samples_len, d_result);

    // Copy the result back to host
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // Check the result
    if (h_result == 0) {
        printf("output = All elements are zero.\n");
    }
    else {
        printf("output = There is at least one element that is not zero.\n");
    }

    hipFree(d_result);

    // Clean up
    hipfftDestroy(segmentPlan);
    hipfftDestroy(IRPlan);
    hipfftDestroy(inversePlan);
    hipFree(sampleSegment);
    hipFree(segmentData);
    hipFree(IRData);
    hipFree(resultData);
}

void copy_from_gpu(float* device_pointer, float* host_pointer, size_t size) {
    CUDA_CHK(hipMemcpy(host_pointer, device_pointer, size, hipMemcpyDeviceToHost));
};

void copy_to_gpu(float* host_pointer, float* device_pointer, size_t size) {
    CUDA_CHK(hipMemcpy(device_pointer, host_pointer, size, hipMemcpyHostToDevice));
};