#include "hip/hip_runtime.h"
#include "./kernels.cuh"
#define CUDA_CHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void CHCK_CUFFT_RES(hipfftResult_t res){
    if (res != 0)
    {
        fprintf(stderr, "CHCK_CUFFT_RES: %d\n", res);
        if (abort) exit(res);
    }
}

inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void fillZeros(float *buf, int size)
{
    int thread = blockDim.x * blockIdx.x + threadIdx.x;
    if (thread < size)
        buf[thread] = 0.f;
}

void fillWithZeroesKernel(float *buf, int size)
{
    int numThreads = 256;
    int numBlocks;
    if (size % numThreads != 0) {
        numBlocks = (size / numThreads) + 1;
    } else {
        numBlocks = size / numThreads;
    }
    fillZeros<<<numBlocks, numThreads>>>(buf, size);
}

__global__ void vectorMultiply(float* a, float* b, float* c, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        c[index] = a[index] * b[index];
    }
}

__global__ void convolute_toeplitz_lower_matrix(float* samples, float* IR, size_t ir_size, float* outputBuffer){
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // where in the "matrix" are we
    int row, col;
    for (int i = 1; i <= ir_size; i++) {
        if (index < i) {
            row = i - 1;
            col = index;
            break;
        }
        index -= i;
    }
    
    /*
    because cell with col = row has I0
    then cell with col = row - 1 has I1 (because we moved column one to the left)
    then cell with col = row - x has Ix
    then row - col = x
    */
    int ir_index = row - col;
    atomicAdd(&outputBuffer[col], samples[row] * IR[ir_index]);
}

__global__ void convolute_toeplitz_lower_matrix_2d(float* samples, float* IR, size_t ir_size, float* outputBuffer) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x <= y && (x < ir_size && y < ir_size)) {
        atomicAdd(&outputBuffer[y*2], samples[x] * IR[y - x]);
        atomicAdd(&outputBuffer[(y*2) - 1], samples[x] * IR[y - x]);
    }
}

__global__ void convolute_toeplitz_vectors(float* samples, float* IR, size_t ir_size, float* outputBuffer){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int samples_offset = index / ir_size;
    int ir_index = index % ir_size;
    /*
    for this part of the convolution samples start at ir_len, thats why we sum ir_len to the samples index
    for each complete multiplication we (the whole IR vector * a subvector of samples) we need to continue with a new subvector
    this new subvector is the samples vector moved one unit forward, thats why we add a samples_offset to the samples index
    */
    atomicAdd(&outputBuffer[ir_size + samples_offset], samples[ir_size + samples_offset] * IR[ir_size - 1 - ir_index]);
}

void convolute_toeplitz_in_gpu(float* samples, float* IR, int ir_len, float* outputBuffer){
    //printf("ir_len: %d", ir_len);
    const int threadsPerBlock = 256;

    // first part, lower matrix multiplication
    /*
    cantidad operaciones = (ir_len * (ir_len + 1)) / 2
    esto es porque en la multiplicacion de la matriz triangular inferior
    la primera fila tiene una operacion, la segunda dos, y la n-esima (hasta el tope que es ir_len)
    equivale a Σ(i) de i=0 a i=n la cual se resuelve con ((n * (n + 1)) / 2)
    */
    int blocksPerGrid;
    if (((ir_len * (ir_len + 1)) / 2) < threadsPerBlock)
        blocksPerGrid = 1;
    else
        blocksPerGrid = (((ir_len * (ir_len + 1)) / 2) / threadsPerBlock) + 1;
    //convolute_toeplitz_lower_matrix<<<blocksPerGrid, threadsPerBlock>>>(samples, IR, ir_len, outputBuffer);
    dim3 threadsPorBlocks(32,32);
    int aaa = (ir_len / 32) + 1;
    dim3 numBlocks(aaa, aaa);
    convolute_toeplitz_lower_matrix_2d << <numBlocks, threadsPorBlocks >> > (samples, IR, ir_len, outputBuffer);

    // second part, vector multiplication
    size_t samples_size = sizeof(samples) / sizeof(float);
    /*
    ya procesamos ir_len celdas del output
    quedan (samples_size - ir_len) celdas restantes
    cada celda tiene ir_len multiplicaciones
    */
    //blocksPerGrid = ((samples_size - ir_len) * ir_len) / threadsPerBlock;
    //convolute_toeplitz_vectors<<<blocksPerGrid, threadsPerBlock>>>(samples, IR, ir_len, outputBuffer); // todo se precisa un offset
}

__global__ void load_complex_vector(hipfftComplex* complex_data, float* real_vector, unsigned int vector_len) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < vector_len){
        complex_data[idx].x = real_vector[idx];
        complex_data[idx].y = 0.0f;
    }
}

__global__ void load_sample_segment(int second, unsigned int sampleRate, unsigned int segment_size_in_seconds, float* segment, float* samples) {
    // loads one second of samples and (segment_size_in_seconds - 1) seconds of zeros
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int sampleIndex = idx + second * sampleRate;

    if (idx < sampleRate * segment_size_in_seconds){
        if (idx < sampleRate) {
            segment[idx] = samples[sampleIndex];
        }
        else {
            segment[idx] = 0.0f;
        }
    }
}

__global__ void multiply_samples_segment_and_ir(int second, unsigned int sampleRate, unsigned int segment_size_in_seconds, hipfftComplex* sampleData, hipfftComplex* IRData) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < sampleRate * segment_size_in_seconds){
        // (a + ib) (c + id) = (ac – bd) + i(ad + bc)
        hipfftComplex temp = sampleData[idx];
        sampleData[idx].x = temp.x * IRData[idx].x - temp.y * IRData[idx].y;
        sampleData[idx].y = temp.x * IRData[idx].y + temp.y * IRData[idx].x;
    }
}

__global__ void add_segment_to_result_buffer(int second, int sampleRate, int segmentLen, float* segment, float* output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // printf("SAMPLE SEGMENT: %d\n", segment);
    if (index < segmentLen) {
        int outputIndex = second * sampleRate + index;
        atomicAdd(&output[outputIndex], segment[index]);
    }
}

__global__ void isAllZeros(float* arr, int size, int* result) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        if (arr[index] != 0.0f) {
            atomicExch(result, 1); // Set result to 1 if any element is not zero
        }
    }
}

// CUDA kernel to check each element if it's zero (complex numbers)
__global__ void isAllZerosComplex(hipfftComplex* arr, int size, int* result) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        if (arr[index].x != 0.0f || arr[index].y != 0.0f) {
            atomicExch(result, 1); // Set result to 1 if any element is not zero
        }
    }
}

bool checkArrayZero(float* IR, unsigned int ir_len) {
    int* d_result; // Device result
    int h_result = 0; // Host result

    // Calculate grid and block sizes
    int blockSize = 256; // Number of threads per block
    int numBlocks = (ir_len + blockSize - 1) / blockSize;
    hipMalloc(&d_result, sizeof(int));
    hipMemcpy(d_result, &h_result, sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel
    isAllZeros << <numBlocks, blockSize >> > (IR, ir_len, d_result);

    // Copy the result back to host
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_result);

    // Check the result
    if (h_result == 0) {
        printf("All elements are zero.\n");
        return true;
    }
    else {
        printf("There is at least one element that is not zero.\n");
        return false;
    }

    
}


__global__ void complexCrossMultiplication(const hipfftDoubleComplex *inputA, const hipfftDoubleComplex *inputB, hipfftDoubleComplex *output, size_t length) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < length) {
        double a = inputA[index].x;
        double b = inputA[index].y;
        double c = inputB[index].x;
        double d = inputB[index].y;

        output[index].x = a * c - b * d;
        output[index].y = a * d + b * c;
    }
}

/*
 * Convoluciona samples con IR, guarda el resultado en outputBuffer
*/
void convoluteFromLiveInput(double* samples, double* IR, unsigned int len, double* outputBuffer) {    
    // Allocate device memory for samples
    hipfftDoubleComplex* segmentData;
    hipMalloc((void**)&segmentData, len * sizeof(hipfftDoubleComplex));
    // Allocate device memory for IR
    hipfftDoubleComplex* IRData;
    hipMalloc((void**)&IRData, len * sizeof(hipfftDoubleComplex));
    
    // Set up FFT plans
    const int batchSize = 1; // Number of batches 
    hipfftHandle plan;
    hipfftPlan1d(&plan, len, HIPFFT_D2Z, batchSize);

    // Convolute and invert result
    hipfftHandle inversePlan;
    hipfftPlan1d(&inversePlan, len, HIPFFT_Z2D, batchSize);
    CHCK_CUFFT_RES(hipfftExecD2Z(plan, IR, IRData));
    CHCK_CUFFT_RES(hipfftExecD2Z(plan, samples, segmentData));

    int blockSize = 256;
    int numBlocks = (len + blockSize - 1) / blockSize;
    complexCrossMultiplication<<<numBlocks, blockSize>>>(segmentData, IRData, segmentData, len);
    hipDeviceSynchronize();
    
    CHCK_CUFFT_RES(hipfftExecZ2D(inversePlan, segmentData, outputBuffer));

    // Clean up
    hipfftDestroy(plan);
    hipfftDestroy(inversePlan);
    hipFree(segmentData);
    hipFree(IRData);
}

void convoluteFromAudioBuffer(float* samples, float* IR, unsigned int samples_len, unsigned int sample_rate, unsigned int ir_len, float* outputBuffer) {
    const int threadsPerBlock = 256;
    int blocks;
    const int batchSize = 1; // Number of batches 
    const int secondsToProcess = samples_len / sample_rate;
    const int segment_size_in_seconds = 2;
    
    // Allocate device memory for samples
    float* sampleSegment;
    int segmentSize = sample_rate * segment_size_in_seconds * sizeof(float);
    hipMalloc((void**)&sampleSegment, segmentSize);
    hipfftComplex* segmentData;
    hipMalloc((void**)&segmentData, sample_rate * segment_size_in_seconds * sizeof(hipfftComplex));
    // Allocate device memory for IR
    hipfftComplex* IRData;
    hipMalloc((void**)&IRData, ir_len * sizeof(hipfftComplex));
    // Set up FFT plans
    hipfftHandle segmentPlan;
    hipfftPlan1d(&segmentPlan, sample_rate * segment_size_in_seconds, HIPFFT_R2C, batchSize);
    hipfftHandle IRPlan;
    hipfftPlan1d(&IRPlan, ir_len, HIPFFT_R2C, batchSize);
    // Invert result
    hipfftHandle inversePlan;
    hipfftPlan1d(&inversePlan, sample_rate * segment_size_in_seconds, HIPFFT_C2R, batchSize);
    // Do FFT on IR, which will be reused a lot
    blocks = (ir_len / threadsPerBlock) + 1;
    hipDeviceSynchronize();
    CHCK_CUFFT_RES(hipfftExecR2C(IRPlan, IR, IRData));

    // Finally convolute, second by second
    /*
    Basically we take each segment and we prolong it with 0's
    Then we do FFT and sum each segment into the total
    Then we invert the total
    Full algorithm can be found on https://www.dspguide.com/ch18/2.htm
    */
    blocks = ((sample_rate * segment_size_in_seconds) / threadsPerBlock) + 1;
    for (int second = 0; second < secondsToProcess; second++) {
        // First second is samples, rest is 0's (this is why we do seconds + 1 as the upper limit)
        load_sample_segment << <blocks, threadsPerBlock >> > (second, sample_rate, segment_size_in_seconds, sampleSegment, samples);
        hipDeviceSynchronize();
        // FFT on the loaded segment, and convolute it with IR
        CHCK_CUFFT_RES(hipfftExecR2C(segmentPlan, sampleSegment, segmentData));
        multiply_samples_segment_and_ir << <blocks, threadsPerBlock >> > (second, sample_rate, segment_size_in_seconds, segmentData, IRData);
        hipDeviceSynchronize();
        // Inverse on the result and save it to buffer
        CHCK_CUFFT_RES(hipfftExecC2R(inversePlan, segmentData, sampleSegment));
        blocks = (segmentSize + threadsPerBlock - 1) / threadsPerBlock;
        add_segment_to_result_buffer<<<blocks, threadsPerBlock>>>(second, sample_rate, sample_rate * segment_size_in_seconds, sampleSegment, outputBuffer);
        hipDeviceSynchronize();
    }
    int* d_result; // Device result
    int h_result = 0; // Host result

    // Allocate memory on the GPU
    hipMalloc(&d_result, sizeof(int));

    // Copy the host array to the device (GPU)
    hipMemcpy(d_result, &h_result, sizeof(int), hipMemcpyHostToDevice);

    // Calculate grid and block sizes
    int blockSize = 256; // Number of threads per block
    int numBlocks = (ir_len + blockSize - 1) / blockSize;

    // Launch the kernel
    isAllZeros << <numBlocks, blockSize >> > (outputBuffer, samples_len, d_result);

    // Copy the result back to host
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // Check the result
    // if (h_result == 0) {
    //     printf("output = All elements are zero.\n");
    // }
    // else {
    //     printf("output = There is at least one element that is not zero.\n");
    // }

    hipFree(d_result);

    // Clean up
    hipfftDestroy(segmentPlan);
    hipfftDestroy(IRPlan);
    hipfftDestroy(inversePlan);
    hipFree(sampleSegment);
    hipFree(segmentData);
    hipFree(IRData);
}

void copy_from_gpu(void* device_pointer, void* host_pointer, size_t size) {
    CUDA_CHK(hipMemcpy(host_pointer, device_pointer, size, hipMemcpyDeviceToHost));
};

void copy_to_gpu(void* host_pointer, void* device_pointer, size_t size) {
    CUDA_CHK(hipMemcpy(device_pointer, host_pointer, size, hipMemcpyHostToDevice));
};

__global__ void d_normalizeBuffers(double * d_outputBuffer_left, double * d_outputBuffer_right, int bufferLength, int value) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (index < bufferLength) {
        d_outputBuffer_left[index] /= value;
        d_outputBuffer_right[index] /= value;
    }
};

void normalizeBuffers(double* d_outputBuffer_left, double* d_outputBuffer_right, int monoBufferLength, int value) {
    int blockSize = 256;
    int numBlocks = (monoBufferLength + blockSize - 1) / blockSize;
    d_normalizeBuffers<<<numBlocks, blockSize>>>(d_outputBuffer_left, d_outputBuffer_right, monoBufferLength, value);
    hipDeviceSynchronize();
};

__global__ void d_zipArrays(const double *arrayA, const double *arrayB, double *outputArray, int length) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (index < length) {
        int outputIndex = index * 2;
        outputArray[outputIndex] = arrayA[index];
        outputArray[outputIndex + 1] = arrayB[index];
    }
};

void zipArrays(double* d_outputBuffer_left, double* d_outputBuffer_right, double* d_outputBuffer, int monoBufferLength) {
    int blockSize = 256;
    int numBlocks = (monoBufferLength + blockSize - 1) / blockSize;
    d_zipArrays<<<numBlocks, blockSize>>>(d_outputBuffer_left, d_outputBuffer_right, d_outputBuffer, monoBufferLength);
    hipDeviceSynchronize();
};

__global__ void d_addDeviceArrayToCircularBuffer(double *deviceArray, size_t dLength, double *circularBuffer, size_t startIndex, size_t hLength) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (index < dLength) {
        size_t circularIndex = (startIndex + index) % hLength;
        // maybe change to atomicAdd?
        circularBuffer[circularIndex] += deviceArray[index];
    }
};

void addDeviceArrayToCircularBuffer(double* deviceArray, int dLength, double *circularBuffer, int startIndex, int hLength){
    int blockSize = 256;
    int numBlocks = (dLength + blockSize - 1) / blockSize;
    d_addDeviceArrayToCircularBuffer<<<numBlocks, blockSize>>>(deviceArray, dLength, circularBuffer, startIndex, hLength);
    hipDeviceSynchronize();
};

__global__ void checkArrayAllZeros(const double* array, bool* result, int length) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < length && array[index] != 0) {
        *result = false; // Array contains a non-zero element
    }
}

__global__ void convertFloatToDouble(const float *input, double *output, size_t length) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < length) {
        output[index] = static_cast<double>(input[index]);
    }
}

void castFloatArrayToDouble(const float *input, double *output, size_t length) {
    // Assuming a block size of 256 threads
    int blockSize = 256;
    int numBlocks = (length + blockSize - 1) / blockSize;

    // Kernel call
    convertFloatToDouble<<<numBlocks, blockSize>>>(input, output, length);
    hipDeviceSynchronize();
}